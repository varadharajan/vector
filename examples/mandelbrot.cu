
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAX_ITER 1000
#define LEFT_MIN -2.5
#define RIGHT_MAX 1
#define BOTTOM_MIN -1
#define TOP_MAX 1

#define IMG_HEIGHT 256
#define IMG_WIDTH 384
#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 16

__global__
void mandelbrot(uchar3 *colors, uchar3 *colorMap,
        float left, float right, float top, float bottom)
{
    size_t xi, yi, xn, yn, i;
    float x0, y0, x, y, xtemp;
    int iter = 0;
    
    xi = threadIdx.x + blockDim.x * blockIdx.x;
    yi = threadIdx.y + blockDim.y * blockIdx.y;
    xn = blockDim.x * gridDim.x;
    yn = blockDim.y * gridDim.y;
    i = xn * yi + xi;

    x0 = left + (right - left) / xn * xi;
    y0 = bottom + (top - bottom) / yn * yi;

    x = y = 0;

    while (iter < MAX_ITER && (x * x + y * y) < 4) {
        xtemp = x * x - y * y + x0;
        y = 2 * x * y + y0;
        x = xtemp;
        iter++;
    }

    colors[i] = colorMap[iter];
}

static void _check(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d\n", file, line);
        fprintf(stderr, "%s\n", hipGetErrorString(err));
        exit(err);
    }
}

#define checkError(err) _check((err), __FILE__, __LINE__)

int write_ppm(const char *fname, uchar3 *colors, 
        unsigned int width, unsigned int height)
{
    FILE *f = fopen(fname, "w");
    size_t x, y, i;
    if (f == NULL) {
        perror("fopen");
        return -1;
    }

    fprintf(f, "P3\n");
    fprintf(f, "%u %u\n", width, height);
    fprintf(f, "255\n");

    for (y = 0; y < height; y++) {
        for (x = 0; x < width; x++) {
            i = y * width + x;
            fprintf(f, "%d %d %d ", colors[i].x, colors[i].y, colors[i].z);
        }
        fprintf(f, "\n");
    }
    return 0;
}

int main(int argc, char *argv[])
{
    float left, right, top, bottom;
    uchar3 colors[IMG_WIDTH * IMG_HEIGHT], colorMap[MAX_ITER + 1];
    uchar3 *d_colors = NULL, *d_colorMap;
    hipError_t err;
    dim3 grid_dim(IMG_WIDTH / BLOCK_WIDTH, IMG_HEIGHT / BLOCK_HEIGHT, 1);
    dim3 block_dim(BLOCK_WIDTH, BLOCK_HEIGHT, 1);
    int i;
    long ran;

    if (argc < 5) {
        fprintf(stderr, "Usage: %s left right top bottom\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    left = atof(argv[1]);
    right = atof(argv[2]);
    top = atof(argv[3]);
    bottom = atof(argv[4]);

    if (left < LEFT_MIN || right > RIGHT_MAX 
            || top > TOP_MAX || bottom < BOTTOM_MIN) {
        fprintf(stderr, "Window out of bounds\n");
        exit(EXIT_FAILURE);
    }

    srandom(time(0));

    for (i = 0; i < MAX_ITER; i++) {
        ran = random();
        colorMap[i].x = (ran >> 16) & 0xff;
        colorMap[i].y = (ran >> 8) & 0xff;
        colorMap[i].z = ran & 0xff;
    }

    colorMap[MAX_ITER].x = 0;
    colorMap[MAX_ITER].y = 0;
    colorMap[MAX_ITER].z = 0;

    err = hipMalloc(&d_colors, sizeof(colors));
    checkError(err);

    err = hipMemset(d_colors, 0, sizeof(colors));

    err = hipMalloc(&d_colorMap, sizeof(colorMap));
    checkError(err);

    err = hipMemcpy(d_colorMap, colorMap, sizeof(colorMap), hipMemcpyHostToDevice);
    checkError(err);

    mandelbrot<<<grid_dim, block_dim>>>(d_colors, d_colorMap, 
            left, right, top, bottom);
    hipDeviceSynchronize();
    checkError(hipGetLastError());

    err = hipMemcpy(colors, d_colors, sizeof(colors), hipMemcpyDeviceToHost);
    checkError(err);

    if (write_ppm("mandelbrot.ppm", colors, IMG_WIDTH, IMG_HEIGHT))
        exit(EXIT_FAILURE);

    hipFree(d_colors);
    hipFree(d_colorMap);

    return 0;
}
